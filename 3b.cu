
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#define BLOCK_SIZE 3
int w=3, h=3;
int size = w*h;
int memsize = sizeof(float)*size;
__global__ void matrixMultiply(float *a, float *b, float *c, int w, int h){
    int tx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int ty = (blockIdx.y * blockDim.y) + threadIdx.y;
    float v = 0;
    int i;
    for(i=0; i<w; i++){
        v+= a[ty*w + i]*b[i*h + tx];
    }
    c[ty*w + tx] = v;
}

void printMatrix(float *a){
    int i;
    for(i=0; i<size; i++){
        printf("%f\t", a[i]);
        if((i+1)%w == 0)
          printf("\n");
    }
}

int main(int argc, char** argv){
    float *ha, *hb, *hc, *a, *b, *c;
    int i;
    ha = (float*)malloc(memsize);
    hb = (float*)malloc(memsize);
    hc = (float*)malloc(memsize);
    
    for(i=0; i<size; i++)
      ha[i] = hb[i] = i;    
    printMatrix(ha);
    printMatrix(hb);
    hipMalloc(&a, memsize);
    hipMalloc(&b, memsize);
    hipMalloc(&c, memsize);
    hipMemcpy(a, ha, memsize, hipMemcpyHostToDevice);
    hipMemcpy(b, hb, memsize, hipMemcpyHostToDevice);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(w/threads.x, h/threads.y);
    matrixMultiply<<<grid,threads>>>(a, b, c, w, w);
    hipMemcpy(hc, c, memsize, hipMemcpyDeviceToHost);
    printMatrix(hc);
    hipFree(a);
    hipFree(b);
    hipFree(c);
    free(ha);
    free(hb);
    free(hc);
    printf("DONE");
    return 0;
}