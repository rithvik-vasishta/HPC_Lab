
#include <hip/hip_runtime.h>
#include<stdio.h>
#define NUM_BLOCKS 8
#define BLOCK_WIDTH 5
__global__ void hello(){
    printf("\nHello from Thread [%d] inside Block [%d]", threadIdx.x, blockIdx.x);
}
int main(){
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
    hipDeviceSynchronize();
    printf("\nDONE");
    return 0;
}