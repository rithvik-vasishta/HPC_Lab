
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
 __global__ void vecAdd(double *a, double *b, double *c, int n){
     int id = blockIdx.x*blockDim.x + threadIdx.x;
     if(id<n){
         c[id] = a[id] + b[id];
     }
 }

 int main(int argc, char** argv){
     int i, n = 100;
     double *ha, *hb, *hc, *a, *b, *c;
     int bytes = sizeof(double)*n;
     ha = (double*)malloc(bytes);
     hb = (double*)malloc(bytes);
     hc = (double*)malloc(bytes);
     hipMalloc(&a, bytes);
     hipMalloc(&b, bytes);
     hipMalloc(&c, bytes);
     for(i=0; i<n; i++){
         ha[i] = hb[i] = rand()%50;
     }
     hipMemcpy(a, ha, bytes, hipMemcpyHostToDevice);
     hipMemcpy(b, hb, bytes, hipMemcpyHostToDevice);
     int blocksize, gridsize;
     blocksize = 1024;
     gridsize = (int)ceil((float)n/blocksize);
     vecAdd<<<gridsize, blocksize>>>(a, b, c, n);
     hipMemcpy(hc, c, bytes, hipMemcpyDeviceToHost);
     for(i=0; i<n; i++){
         printf("%f + %f = %f\n", ha[i], hb[i], hc[i]);
     }
     printf("\nDONE\n");
     return 0;
 }